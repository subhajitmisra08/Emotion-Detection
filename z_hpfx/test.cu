#include <iostream>
#include <hip/hip_runtime.h>
#include<cudnn.h>

__global__ void hello() {
    printf("Hello, CUDA!\n");
}

int main() {
    hello<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}